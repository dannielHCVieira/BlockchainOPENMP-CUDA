#include "hip/hip_runtime.h"
//
// Created by Dave Nash on 20/10/2017.
//

#include "../../headers/cuda/Block.cuh"
#include "../../headers/cuda/sha256_CPU.cuh"
#include "../../headers/cuda/sha256_GPU.cuh"

#define SOLUTION_LEN 25

__constant__ unsigned char setOfCharacter[63] = { "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ1234567890" };

Block::Block(uint32_t nIndexIn, const string &sDataIn) : _nIndex(nIndexIn), _sData(sDataIn)
{
    _sNonce = '0';
    _tTime = time(nullptr);

    sHash = _CalculateHash();
}

__global__ void createSolutionChecker(bool* block_isSolved){
    *block_isSolved = false;
}


__device__ unsigned long long generateRngSeed_GPU(unsigned long long x)
{
    x ^= (x << 21);
    x ^= (x >> 35);
    x ^= (x << 4);
    return x;
}


__global__ void SHA256_CUDA(unsigned char* input_string,unsigned char* solution, bool* block_isSolved,uint32_t nDifficulty,unsigned long long seed, size_t text_len){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long newSeed = seed;
    newSeed = (unsigned long long) i + newSeed;

    unsigned char digest[32], random[SOLUTION_LEN];

    memset(digest, 0, SHA256::DIGEST_SIZE);
    memset(random, 0, SOLUTION_LEN);


    for(int j = 0; j < SOLUTION_LEN; j++){
        newSeed = generateRngSeed_GPU(newSeed);
        int randomIdx = (int)(newSeed % 62);
        random[j] = setOfCharacter[randomIdx];
    }

    SHA256GPU::sha256(input_string, random, text_len, SOLUTION_LEN, digest);

    for (int j = 0; j < nDifficulty; j++){
        if(digest[j] > 0){
            return;
        }
    }

    if(*block_isSolved){
        return;
    }

    *block_isSolved = true;
    
    for(int j = 0; j < SOLUTION_LEN; j++){
        solution[j] = random[j];
    }
}

void Block::generateRngSeed_CPU(unsigned long long* x)
{
    *x ^= (*x << 21);
    *x ^= (*x >> 35);
    *x ^= (*x << 4);
}

// OpenMP (GPU)
void Block::MineBlock(uint32_t nDifficulty)
{

    /**
     * @brief 
     * Para paralelizar o código da Blockchain utilizando CUDA na GPU, precisamos localizar a parte mais pesada do programa, que é 
     * justamente a mineração do bloco, mais especificamente no calculo do hash SHA256 que é feito até chegar em uma solução. Por isso,
     * vamos paralelizar esta parte. Para isso, a ideia que temos é de alocar um bloco na GPU para receber uma string que será utilizada 
     * na criptografia, um bloco onde estará contido o resultado da criptografia e um bloco para um valor booleano que será utilizado para 
     * checar se o resultado bate com a string original. Após encontrar a solução em GPU, que deve ser mais ágil que em CPU, devolveremos a 
     * resposta para a CPU que ira utilizar para criar o novo bloco.
     */

    uint32_t dimGrid = 1500, dimBlock = 256;

    stringstream str_stream;
    str_stream << _nIndex << _tTime << sPrevHash;

    string str_stream_str = str_stream.str();

    unsigned char* input_string = (unsigned char*)str_stream_str.c_str();
    unsigned char* d_input;

    unsigned char* block_solution = (unsigned char*)malloc(sizeof(char) * SOLUTION_LEN);
    unsigned char* d_solution;

    bool* block_isSolved = (bool*)malloc(sizeof(bool));
    bool* d_isSolved;

    //Alocação da memoria em GPU e copia para da string de input para a mesma
    hipMalloc(&d_input, sizeof(char) * str_stream_str.length());
    hipMemcpy(d_input, input_string, sizeof(char) * str_stream_str.length(), hipMemcpyHostToDevice);
    
    //Alocação da memoria em GPU e copia da string que conterá a solução
    hipMalloc(&d_solution, sizeof(char) * SOLUTION_LEN);

    //Alocação da memoria em GPU para o booleano
    hipMalloc(&d_isSolved, sizeof(bool));

    //Seta o solução checker como falso inicialmente
    unsigned long long seed = static_cast<unsigned long long>(time(nullptr));
    createSolutionChecker<<<1,1>>>(d_isSolved);

    //Usa o SHA256 paralelizado para chegar na solução
    bool solution = false;
    while(!solution) {
        generateRngSeed_CPU(&seed);

        SHA256_CUDA<<<dimGrid, dimBlock>>>(d_input, d_solution, d_isSolved, nDifficulty, seed, str_stream_str.length());
        
        hipDeviceSynchronize();

        hipMemcpy(block_isSolved, d_isSolved, sizeof(int), hipMemcpyDeviceToHost);

        if(*block_isSolved){
            hipMemcpy(block_solution, d_solution, sizeof(char) * SOLUTION_LEN, hipMemcpyDeviceToHost);
            solution = true;

            break;
        }
    }

    hipDeviceReset();

    _sNonce = string((const char*)block_solution);
    sHash = _CalculateHash();
    
    cout << "Block mined: " << sHash << endl;
}

inline string Block::_CalculateHash() const
{
    stringstream ss;
    ss << _nIndex << sPrevHash << _tTime << _sData << _sNonce;
    
    return sha256(ss.str());
}


